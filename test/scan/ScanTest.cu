/* Copyright STIFTELSEN SINTEF 2010
 *
 * This file is part of the SIST Library.
 *
 * Author(s): Christopher Dyken, <christopher.dyken@sintef.no>
 *            Johan Seland, <johan.seland@sintef.no>
 *
 * SIST is free software: you can redistribute it and/or modify it under the
 * terms of the GNU General Public License as published by the Free Software
 * Foundation, either version 3 of the License, or (at your option) any later
 * version.
 *
 * HPMC is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
 * A PARTICULAR PURPOSE.  See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along with
 * SIST.  If not, see <http://www.gnu.org/licenses/>.
 */
#include <iostream>
#include <vector>
#include <cstdlib>
#include <cmath>
#include <string>
#include <algorithm>
#include <functional>
#include <numeric>
#include <typeinfo>
#include <map>
#include <hip/hip_runtime.h>
#include <sist/scan/scan.hpp>
#include <cudpp.h>
#include <thrust/scan.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#define BOOST_TEST

#ifdef BOOST_TEST
#define BOOST_TEST_DYN_LINK
#define BOOST_TEST_MODULE ScanTest

#include <boost/test/unit_test.hpp>
#include <boost/test/parameterized_test.hpp>
#include <boost/test/test_case_template.hpp>
#include <boost/mpl/list.hpp>

#endif // BOOST_TEST

/** This file defines tests for the various scan-implemenations in SIST and also allows for
     benchmarking against Thrust and CUDPP, the latter only in release mode on Windows.

     The various scan-algorithms are called by overriding AbstractScanBenchmark::doScan below.
     The testing of results are done in BOOST_FIXTURE_TEST cases at the bottom of the file. 

     Due to the design of Boost test, one must pass --log_level=message as input to the program to see
     timings.
*/

/** Helper macro that checks for CUDA errors, and exits if any. */
#define CHECK_CUDA do {                                                        \
    hipError_t res = hipGetLastError();                                      \
    if( res != hipSuccess ) {                                                 \
        std::cerr << __FILE__ << '@'                                           \
                  << __LINE__ << ": CUDA error: "                              \
                  << hipGetErrorString( res )                                 \
                  << std::endl;                                                \
        hipDeviceReset();                                                      \
        exit( EXIT_FAILURE );                                                  \
    }                                                                          \
} while(0)

/** ScanFixture allocates GPU and CPUto be used by all the various scan implemntations.
    It populates the input with random data and computes the sum of the input to be used 
    by all the tests for validation.
*/
struct ScanFixture {

    ScanFixture( unsigned int numElements = 1024u ) : N( numElements ) {
        CHECK_CUDA;
        input.resize( N );
        output.resize( input.size() + 1 );

        srand( 42 );
        for(size_t i=0; i<input.size(); i++ ) {
            input[i] = rand() & 0xffu;        
        }

        sum_golden = std::accumulate( begin(input), end(input), 0 );
        
        hipMalloc( &input_d, sizeof(unsigned int)*(input.size()) );
        hipMalloc( &scratch_d, sist::scan::scratchBufferBytesize( input.size() ) );
        hipMalloc( &output_d, sizeof(unsigned int)*(output.size()) );
        hipMemset( output_d, ~0u, sizeof(unsigned int)*output.size() );

        hipMemcpy( input_d, input.data(), sizeof(unsigned int)*input.size(), hipMemcpyHostToDevice );

        CHECK_CUDA;
    }

    ~ScanFixture() {
        CHECK_CUDA;
        hipFree( input_d );
        hipFree( output_d );
        hipFree( scratch_d );
        CHECK_CUDA;
    }

    const unsigned int N;
    std::vector<unsigned int> input;
    std::vector<unsigned int> output;  

    unsigned int* input_d;
    unsigned int* output_d;
    unsigned int* scratch_d;

    unsigned int sum_golden;
};
    

/** AbstractScanBenchmark is responsible for priming (warmup) and execution of the various scan algorithms. 
    The concrete call to a given scan implemenation is handled by overriding the doScan-method. */
class AbstractScanBenchmark {
public:
    AbstractScanBenchmark( 
        unsigned int* input_d, unsigned int* output_d, unsigned int* scratch_d,
        const std::vector<unsigned int>& input, std::vector<unsigned int>& output ) 
        : 
        input( input ),
        output( output ), 
        input_d( input_d ),
        output_d( output_d ),
        scratch_d( scratch_d ),
        its( 100 )
    {
        hipEventCreate( &start );
        hipEventCreate( &stop );

        CHECK_CUDA;
    }

    ~AbstractScanBenchmark() {
        hipEventDestroy( start );
        hipEventDestroy( stop );
    }

    virtual void benchmarkScan( size_t N, float ref ) {
        ms = 0.0f;
        // WarmUp
        for(int i=0; i<(its+9)/10; i++) {
            doScan( N );
        }

        // Benchmark
        hipEventRecord( start );
        for( int i = 0; i < its; ++i ) {
              doScan( N );
        }
        hipEventRecord( stop );
        hipMemcpy( output.data(), output_d, sizeof(unsigned int)*(output.size()), hipMemcpyDeviceToHost  );
        hipEventSynchronize( stop );

        hipEventElapsedTime( &ms, start, stop );        
        //BOOST_TEST_MESSAGE( "Time: " << ms/its << "ms" );
    }
    
    virtual void doScan( size_t N ) = 0;

    float ms;
    const int its;

    hipEvent_t start;
    hipEvent_t stop;
    const std::vector<unsigned int>& input;
    std::vector<unsigned int>& output;
    unsigned int* input_d;
    unsigned int* output_d;
    unsigned int* scratch_d;
};

template<class T>
void check_exclusive_scan_result( const T& input, const T& output, const size_t N ) {
    unsigned int sum = 0;
    for( size_t i = 0; i < N; i++ ) {
        BOOST_CHECK_EQUAL( output[i], sum );
        sum += input[i];
    }
}

template<class T>
void check_inclusive_scan_result( const T& input, const T& output, const size_t N ) {
    unsigned int sum = 0;
    for( size_t i = 0; i < N; i++ ) {
        sum += input[i];
        BOOST_CHECK_EQUAL( output[i], sum );        
    }
}


class BenchmarkThrustExclusiveScan : public AbstractScanBenchmark {
public:
    BenchmarkThrustExclusiveScan(
        unsigned int* input_d, unsigned int* output_d, unsigned int* scratch_d,
        const std::vector<unsigned int>& input, std::vector<unsigned int>& output  )
        :
          AbstractScanBenchmark( input_d, output_d, scratch_d, input, output ),
          input_d( thrust::device_pointer_cast( input_d ) ),
          output_d( thrust::device_pointer_cast( output_d ) )
    {         
    }

    void doScan( size_t N ) {
        thrust::exclusive_scan( input_d, input_d + N, output_d );
    }        
private:    
    thrust::device_ptr<unsigned int> input_d;
    thrust::device_ptr<unsigned int> output_d;
};

class BenchmarkCUDPPExclusiveScan : public AbstractScanBenchmark {
public:
    BenchmarkCUDPPExclusiveScan( unsigned int* input_d, unsigned int* output_d, unsigned int* scratch_d,
                                 const std::vector<unsigned int>& input, std::vector<unsigned int>& output  ) 
        : AbstractScanBenchmark( input_d, output_d, scratch_d, input, output ),
          scanplan( 0 )
    {        
        cudppCreate( &cudpp_handle );

        cudpp_config.op 		= CUDPP_ADD;
        cudpp_config.datatype 	= CUDPP_UINT;
        cudpp_config.algorithm 	= CUDPP_SCAN;
        cudpp_config.options	= CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;

        cudpp_res = cudppPlan( cudpp_handle, &scanplan, cudpp_config, input.size(), 1, 0 );
            
        if( cudpp_res != CUDPP_SUCCESS ) {            
            throw std::runtime_error( "CUDPP Failed to init. " );
       }
    }

    ~BenchmarkCUDPPExclusiveScan() {
        cudppDestroy( cudpp_handle );
    }


    void doScan( size_t N ) {
        cudpp_res = cudppScan( scanplan, output_d, input_d, N );
        if( cudpp_res != CUDPP_SUCCESS ) {
            throw std::runtime_error( "CUDPP failed during scan. ");
        }
    }

    CUDPPHandle cudpp_handle;    
    CUDPPConfiguration cudpp_config;            
    CUDPPHandle scanplan;
    CUDPPResult cudpp_res;
};

class BenchmarkSistExclusiveScan : public AbstractScanBenchmark {
public:
     BenchmarkSistExclusiveScan( unsigned int* input_d, unsigned int* output_d, unsigned int* scratch_d,
                                const std::vector<unsigned int>& input, std::vector<unsigned int>& output  ) 
        : AbstractScanBenchmark( input_d, output_d, scratch_d, input, output )
    {}

     void doScan( size_t N ) {
         sist::scan::exclusiveScan( output_d, scratch_d, input_d, N );
     }
};

class BenchmarkSistInclusiveScan : public AbstractScanBenchmark {
public:
    BenchmarkSistInclusiveScan( unsigned int* input_d, unsigned int* output_d, unsigned int* scratch_d,
                                const std::vector<unsigned int>& input, std::vector<unsigned int>& output  ) 
        : AbstractScanBenchmark( input_d, output_d, scratch_d, input, output )
    {}

    void doScan( size_t N ) {
         sist::scan::inclusiveScan( output_d, scratch_d, input_d, N );
    }    
};

class BenchmarkSistInclusiveScanWithSum : public AbstractScanBenchmark {
public:
    BenchmarkSistInclusiveScanWithSum( unsigned int* input_d, unsigned int* output_d, unsigned int* scratch_d,
                                const std::vector<unsigned int>& input, std::vector<unsigned int>& output  ) 
        : AbstractScanBenchmark( input_d, output_d, scratch_d, input, output )
    {         
        hipHostAlloc( &zerocopy, sizeof(unsigned int), hipHostMallocMapped );
        hipHostGetDevicePointer( &zerocopy_d, zerocopy, 0 );
        CHECK_CUDA;

        *zerocopy = 42;
    }

    ~BenchmarkSistInclusiveScanWithSum() {
        hipHostFree( zerocopy );
    }

    void doScan( size_t N ) {
        sist::scan::inclusiveScanWriteSum( output_d, zerocopy_d, scratch_d, input_d, N );
    }
    
    unsigned int* zerocopy;
    unsigned int* zerocopy_d;
};

class BenchmarkSistExclusiveScanWithSum : public AbstractScanBenchmark {
public:
    BenchmarkSistExclusiveScanWithSum( unsigned int* input_d, unsigned int* output_d, unsigned int* scratch_d,
                                const std::vector<unsigned int>& input, std::vector<unsigned int>& output  ) 
        : AbstractScanBenchmark( input_d, output_d, scratch_d, input, output )
    {         
        hipHostAlloc( &zerocopy, sizeof(unsigned int), hipHostMallocMapped );
        hipHostGetDevicePointer( &zerocopy_d, zerocopy, 0 );
        CHECK_CUDA;

        *zerocopy = 42;
    }

    ~BenchmarkSistExclusiveScanWithSum() {
        hipHostFree( zerocopy );
    }

    void doScan( size_t N ) {
        sist::scan::exclusiveScanWriteSum( output_d, zerocopy_d, scratch_d, input_d, N );
    }
    
    unsigned int* zerocopy;
    unsigned int* zerocopy_d;
};

class BenchmarkSistExclusiveScanPadWithSum : public AbstractScanBenchmark {
public:
    BenchmarkSistExclusiveScanPadWithSum( unsigned int* input_d, unsigned int* output_d, unsigned int* scratch_d,
                                const std::vector<unsigned int>& input, std::vector<unsigned int>& output  ) 
        : AbstractScanBenchmark( input_d, output_d, scratch_d, input, output )
    {}


    void doScan( size_t N ) {
        sist::scan::exclusiveScanPadWithSum( output_d, scratch_d, input_d, N );
    }    
};

class BenchmarkSistExclusiveScanPadWithSumWriteSum : public AbstractScanBenchmark {
public:
    BenchmarkSistExclusiveScanPadWithSumWriteSum( unsigned int* input_d, unsigned int* output_d, unsigned int* scratch_d,
                                const std::vector<unsigned int>& input, std::vector<unsigned int>& output  ) 
        : AbstractScanBenchmark( input_d, output_d, scratch_d, input, output )
    {
        hipHostAlloc( &zerocopy, sizeof(unsigned int), hipHostMallocMapped );
        hipHostGetDevicePointer( &zerocopy_d, zerocopy, 0 );
        CHECK_CUDA;

        *zerocopy = 42;
    }

    ~BenchmarkSistExclusiveScanPadWithSumWriteSum() {
        hipHostFree( zerocopy );
    }

    void doScan( size_t N ) {
        sist::scan::exclusiveScanPadWithSumWriteSum( output_d, zerocopy_d, scratch_d, input_d, N );
    }    

    unsigned int* zerocopy;
    unsigned int* zerocopy_d;
};

#ifdef BOOST_TEST
BOOST_FIXTURE_TEST_CASE( BenchMarkThrustExclusiveScan, ScanFixture ) {
    BenchmarkThrustExclusiveScan bench( input_d, output_d, scratch_d, input, output );
    bench.benchmarkScan( input.size(), 0.0f );
        
    check_exclusive_scan_result( input, output, N ); 
}

BOOST_FIXTURE_TEST_CASE( BenchMarkCUDPPExclusiveScan, ScanFixture ) {
    BenchmarkCUDPPExclusiveScan bench( input_d, output_d, scratch_d, input, output );
    bench.benchmarkScan( input.size(), 0.0f );
        
    check_exclusive_scan_result( input, output, N ); 
}

BOOST_FIXTURE_TEST_CASE( BenchMarkSistExclusiveScan, ScanFixture ) {
    BenchmarkSistExclusiveScan bench(  input_d, output_d, scratch_d, input, output );
    bench.benchmarkScan( input.size(), 0.0f );
    
    check_exclusive_scan_result( input, output, N );
    BOOST_CHECK_EQUAL( ~0u, bench.output[N ]);
}

BOOST_FIXTURE_TEST_CASE( BenchMarkSistInclusiveScan, ScanFixture ) {
    BenchmarkSistInclusiveScan bench(  input_d, output_d, scratch_d, input, output );
    bench.benchmarkScan( input.size(), 0.0f );
    
    check_inclusive_scan_result( input, output, N );    
    BOOST_CHECK_EQUAL( ~0u, bench.output[N] );
}


BOOST_FIXTURE_TEST_CASE( BenchMarkSistInclusiveScanWithSum, ScanFixture ) {
    BenchmarkSistInclusiveScanWithSum bench(  input_d, output_d, scratch_d, input, output );
    bench.benchmarkScan( input.size(), 0.0f );
        
    check_inclusive_scan_result( input, output, N );
    BOOST_CHECK_EQUAL( ~0u, bench.output[N] );
    BOOST_CHECK_EQUAL( sum_golden, *(bench.zerocopy) );    
}

BOOST_FIXTURE_TEST_CASE( BenchMarkSistExclusiveScanWithSum, ScanFixture ) {
    BenchmarkSistExclusiveScanWithSum bench(  input_d, output_d, scratch_d, input, output );
    bench.benchmarkScan( input.size(), 0.0f );
        
    check_exclusive_scan_result( input, output, N );    
    BOOST_CHECK_EQUAL( ~0u, bench.output[N] );
    BOOST_CHECK_EQUAL( sum_golden, *(bench.zerocopy) );    
}

BOOST_FIXTURE_TEST_CASE( BenchMarkSistExclusiveScanPadWithSum, ScanFixture ) {
    BenchmarkSistExclusiveScanPadWithSum bench(  input_d, output_d, scratch_d, input, output );
    bench.benchmarkScan( input.size(), 0.0f );
        
    check_exclusive_scan_result( input, output, N );    
    BOOST_CHECK_EQUAL( sum_golden, bench.output[N] );    
}

BOOST_FIXTURE_TEST_CASE( BenchMarkSistExclusiveScanPadWithSumWriteSum, ScanFixture ) {
    BenchmarkSistExclusiveScanPadWithSumWriteSum bench(  input_d, output_d, scratch_d, input, output );
    bench.benchmarkScan( input.size(), 0.0f );
        
    check_exclusive_scan_result( input, output, N );    
    BOOST_CHECK_EQUAL( sum_golden, bench.output[N] );
    BOOST_CHECK_EQUAL( sum_golden, *(bench.zerocopy) );    
}


// The first element in this list is used as the reference performance in the PerformanceTest
typedef boost::mpl::list<
                         BenchmarkCUDPPExclusiveScan,
                         BenchmarkThrustExclusiveScan, 
                         BenchmarkSistInclusiveScan, BenchmarkSistInclusiveScanWithSum,
                         BenchmarkSistExclusiveScan, BenchmarkSistExclusiveScanPadWithSum, BenchmarkSistExclusiveScanPadWithSumWriteSum>
    testTypes;

std::map<unsigned int, double> referencePerformance;

BOOST_AUTO_TEST_CASE_TEMPLATE( PerformanceTest, T, testTypes ) 
{
    const int maxN = 0x1 << 20;    
    BOOST_MESSAGE( typeid(T).name() );
    for(int N = maxN; N>0; N = N/2.15 ) {    
        ScanFixture s(  N ); 

        T scan( s.input_d, s.output_d, s.scratch_d, s.input, s.output );
        scan.benchmarkScan( s.input.size(), 0.0f );

        auto performance = scan.ms/scan.its;
        auto speedup = 1.0;
        auto it = referencePerformance.find( N );

        if ( it == referencePerformance.end() ) { // Establish reference performance
            referencePerformance[N] = performance;
        } else { // compute speedup relative to the reference
            speedup = it->second/performance;
        }

        BOOST_MESSAGE( " size=" << N << "\t time=" << performance << " ms\t speedup=" << speedup  );
    }
}



int old_main( int argc, char** argv )
#else
int main( int argc, char** argv )
#endif // BOOST_TEST
{
    int cuda_device = 0;

    for( int i=1; i<argc; i++ ) {
        std::string arg( argv[i] );
        if( (arg == "-d") && (i+1 < argc) ) {
            cuda_device = atoi( argv[i+1] );
        }

    }

    int device_count;
    hipGetDeviceCount( &device_count );
    if( device_count == 0 ) {
        std::cerr << "No CUDA devices present, exiting." << std::endl;
        return -1;
    }
    for(int dev=0; dev<device_count; dev++ ) {
        hipDeviceProp_t dev_prop;
        hipGetDeviceProperties( &dev_prop, dev );
        std::cerr << "CUDA device " << dev << ": "
                  << dev_prop.name << ", "
                  << "compute cap " << dev_prop.major << "."  << dev_prop.minor
                  << std::endl;
    }
    if( (cuda_device < 0 ) || (device_count <= cuda_device) ) {
        std::cerr << "Illegal device " << cuda_device << ", exiting." << std::endl;
        return -1;
    }
    std::cerr << "Using cuda device " << cuda_device << std::endl;
    hipSetDevice( cuda_device );


    //std::vector<unsigned int> input( 0x03fffc00 );
    std::vector<unsigned int> input( 16384 );
    std::vector<unsigned int> output( input.size()+1 );

    unsigned int* input_d;
    hipMalloc( &input_d, sizeof(unsigned int)*(input.size()) );


    unsigned int* scratch_d;
    hipMalloc( &scratch_d, sist::scan::scratchBufferBytesize( input.size() ) );

    unsigned int* output_d;
    hipMalloc( &output_d, sizeof(unsigned int)*(output.size()) );

    CHECK_CUDA;

    CUDPPHandle cudpp_handle;
    cudppCreate( &cudpp_handle );




#if 1
    for(int N=input.size(); N>0; N = N/2.15 ) {
#else
    if( 1 ) {
        int N=743;
#endif
        std::cerr << "N=" << N << "\n";
        //std::default_random_engine dre;
        //std::uniform_int_distribution<unsigned int> di( 0, 255 );
        //auto dice = std::bind( di, dre );

        //std::generate( input.begin(), input.end(), dice );

        srand( 42 );
        for(size_t i=0; i<input.size(); i++ ) {
            input[i] = rand() & 0xffu;
        }

        hipMemcpy( input_d, input.data(), sizeof(unsigned int)*input.size(), hipMemcpyHostToDevice );

        float ref;
        if(1) {
            float ms;
            int its = 100;
            hipEvent_t start, stop;
            hipEventCreate( &start );
            hipEventCreate( &stop );
            hipMemset( output_d, ~0u, sizeof(unsigned int)*output.size() );

            CUDPPConfiguration cudpp_config;
            cudpp_config.op 		= CUDPP_ADD;
            cudpp_config.datatype 	= CUDPP_UINT;
            cudpp_config.algorithm 	= CUDPP_SCAN;
            cudpp_config.options	= CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;

            CUDPPHandle scanplan = 0;
            CUDPPResult cudpp_res = cudppPlan( cudpp_handle, &scanplan, cudpp_config, N, 1, 0 );
            if( cudpp_res != CUDPP_SUCCESS ) {
                std::cerr << "CUDPP Failed to init, exiting.\n";
                exit( EXIT_FAILURE );
            }

            for(int i=0; i<(its+9)/10; i++) {
                cudpp_res = cudppScan( scanplan, output_d, input_d, N );
                if( cudpp_res != CUDPP_SUCCESS ) {
                    std::cerr << "CUDPP Failed in warmup.\n";
                }
            }

            hipEventRecord( start );
            for(int i=0; i<its; i++) {
                cudpp_res = cudppScan( scanplan, output_d, input_d, N );
                if( cudpp_res != CUDPP_SUCCESS ) {
                    std::cerr << "CUDPP Failed in warmup.\n";
                    exit( EXIT_FAILURE );
                }
            }
            hipEventRecord( stop );
            hipEventSynchronize( stop );
            hipMemcpy( output.data(), output_d, sizeof(unsigned int)*(output.size()), hipMemcpyDeviceToHost  );
            hipEventElapsedTime( &ms, start, stop );
            CHECK_CUDA;

            int fails = 0;
            unsigned int sum = 0;
            for(int i=0; i<N; i++ ) {
                if( output[i] != sum ) {
                    fails++;
                }
                sum += input[i];
            }
            std::cerr << "\tCUDPP\tE="<< fails
                      << "\t"
                      << "\t"
                      << "\ttime=" << (ms/its) << "ms.\n";
            ref = ms/its;
        }

        BenchmarkThrustExclusiveScan thrustScan( input_d, output_d, scratch_d, input, output );
        thrustScan.benchmarkScan( N, ref );

        // inclusive scan
        if(1) {
            float ms;
            int its = 100;
            hipEvent_t start, stop;
            hipEventCreate( &start );
            hipEventCreate( &stop );
            hipMemset( output_d, ~0u, sizeof(unsigned int)*output.size() );
            for(int i=0; i<(its+9)/10; i++) {
                sist::scan::inclusiveScan( output_d,
                                     scratch_d,
                                     input_d,
                                     N );
            }
            hipEventRecord( start, 0 );
            for(int i=0; i<its; i++) {
                sist::scan::inclusiveScan( output_d,
                                     scratch_d,
                                     input_d,
                                     N );
            }
            hipEventRecord( stop );
            hipEventSynchronize( stop );
            hipMemcpy( output.data(), output_d, sizeof(unsigned int)*(output.size()), hipMemcpyDeviceToHost  );
            hipEventElapsedTime( &ms, start, stop );
            CHECK_CUDA;

            int fails = 0;
            unsigned int sum = 0;
            for(int i=0; i<N; i++ ) {
                sum += input[i];
                if( output[i] != sum ) {
                    fails++;
                }
            }
            std::cerr << "\tin\tE="<< fails
                      << "\tS=" << (output[N]==~0u?"ok":"ERR" )
                      << "\t"
                      << "\ttime=" << (ms/its) << "ms"
                      << "\tspeedup=" << (ref/(ms/its)) <<"X.\n";
        }

        // inclusive scan with sum
        if(1) {
            unsigned int* zerocopy;
            hipHostAlloc( &zerocopy, sizeof(unsigned int), hipHostMallocMapped );

            unsigned int* zerocopy_d;
            hipHostGetDevicePointer( &zerocopy_d, zerocopy, 0 );
            CHECK_CUDA;

            *zerocopy = 42;

            float ms;
            int its = 100;
            hipEvent_t start, stop;
            hipEventCreate( &start );
            hipEventCreate( &stop );
            hipMemset( output_d, ~0u, sizeof(unsigned int)*output.size() );
            for(int i=0; i<(its+9)/10; i++) {
                sist::scan::inclusiveScanWriteSum( output_d,
                                             zerocopy_d,
                                             scratch_d,
                                             input_d,
                                             N );
            }
            hipEventRecord( start, 0 );
            for(int i=0; i<its; i++) {
                sist::scan::inclusiveScanWriteSum( output_d,
                                             zerocopy_d,
                                             scratch_d,
                                             input_d,
                                             N );
            }
            hipEventRecord( stop );
            hipEventSynchronize( stop );
            hipMemcpy( output.data(), output_d, sizeof(unsigned int)*(output.size()), hipMemcpyDeviceToHost  );
            hipEventElapsedTime( &ms, start, stop );
            CHECK_CUDA;

            int fails = 0;
            unsigned int sum = 0;
            for(int i=0; i<N; i++ ) {
                sum += input[i];
                if( output[i] != sum ) {
                    fails++;
                }
            }
            std::cerr << "\tin+S\tE="<< fails
                      << "\tS=" << (output[N]==~0u?"ok":"ERR" )
                      << "\tZ=" << ((*zerocopy == sum) ? "ok":"ERR" )
                      << "\ttime=" << (ms/its) << "ms"
                      << "\tspeedup=" << (ref/(ms/its)) <<"X.\n";
            hipHostFree( zerocopy );
        }

        // exclusive scan
        if(1) {
            float ms;
            int its = 100;
            hipEvent_t start, stop;
            hipEventCreate( &start );
            hipEventCreate( &stop );
            hipMemset( output_d, ~0u, sizeof(unsigned int)*output.size() );
            for(int i=0; i<(its+9)/10; i++) {
                sist::scan::exclusiveScan( output_d,
                                     scratch_d,
                                     input_d,
                                     N );
            }
            hipEventRecord( start, 0 );
            for(int i=0; i<its; i++) {
                sist::scan::exclusiveScan( output_d,
                                     scratch_d,
                                     input_d,
                                     N );
            }
            hipEventRecord( stop );
            hipEventSynchronize( stop );
            hipMemcpy( output.data(), output_d, sizeof(unsigned int)*(output.size()), hipMemcpyDeviceToHost  );
            hipEventElapsedTime( &ms, start, stop );
            CHECK_CUDA;

            int fails = 0;
            unsigned int sum = 0;
            for(int i=0; i<N; i++ ) {
                if( output[i] != sum ) {
                    fails++;
                }
                sum += input[i];
            }
            std::cerr << "\tex\tE="<< fails
                      << "\tS=" << (output[N]==~0u?"ok":"ERR" )
                      << "\t"
                      << "\ttime=" << (ms/its) << "ms"
                      << "\tspeedup=" << (ref/(ms/its)) <<"X.\n";
        }


        // exclusive scan with sum
        if(1) {
            unsigned int* zerocopy;
            hipHostAlloc( &zerocopy, sizeof(unsigned int), hipHostMallocMapped );

            unsigned int* zerocopy_d;
            hipHostGetDevicePointer( &zerocopy_d, zerocopy, 0 );
            CHECK_CUDA;

            *zerocopy = 42;

            float ms;
            int its = 100;
            hipEvent_t start, stop;
            hipEventCreate( &start );
            hipEventCreate( &stop );
            hipMemset( output_d, ~0u, sizeof(unsigned int)*output.size() );
            for(int i=0; i<(its+9)/10; i++) {
                sist::scan::exclusiveScanWriteSum( output_d,
                                             zerocopy_d,
                                             scratch_d,
                                             input_d,
                                             N );
            }
            hipEventRecord( start, 0 );
            for(int i=0; i<its; i++) {
                sist::scan::exclusiveScanWriteSum( output_d,
                                             zerocopy_d,
                                             scratch_d,
                                             input_d,
                                             N );
            }
            hipEventRecord( stop );
            hipEventSynchronize( stop );
            hipMemcpy( output.data(), output_d, sizeof(unsigned int)*(output.size()), hipMemcpyDeviceToHost  );
            hipEventElapsedTime( &ms, start, stop );
            CHECK_CUDA;

            int fails = 0;
            unsigned int sum = 0;
            for(int i=0; i<N; i++ ) {
                if( output[i] != sum ) {
                    fails++;
                }
                sum += input[i];
            }
            std::cerr << "\tex+S\tE="<< fails
                      << "\tS=" << (output[N]==~0u?"ok":"ERR" )
                      << "\tZ=" << ((*zerocopy == sum) ? "ok":"ERR" )
                      << "\ttime=" << (ms/its) << "ms"
                      << "\tspeedup=" << (ref/(ms/its)) <<"X.\n";
            hipHostFree( zerocopy );
        }

        // exlusive with padded sum
        if(1) {
            float ms;
            int its = 100;
            hipEvent_t start, stop;
            hipEventCreate( &start );
            hipEventCreate( &stop );
            hipMemset( output_d, ~0u, sizeof(unsigned int)*output.size() );
            for(int i=0; i<(its+9)/10; i++) {
                sist::scan::exclusiveScanPadWithSum( output_d,
                                           scratch_d,
                                           input_d,
                                           N );
            }
            hipEventRecord( start, 0 );
            for(int i=0; i<its; i++) {
                sist::scan::exclusiveScanPadWithSum( output_d,
                                               scratch_d,
                                               input_d,
                                               N );
            }
            hipEventRecord( stop );
            hipEventSynchronize( stop );
            hipMemcpy( output.data(), output_d, sizeof(unsigned int)*(output.size()), hipMemcpyDeviceToHost  );
            hipEventElapsedTime( &ms, start, stop );
            CHECK_CUDA;

            int fails = 0;
            unsigned int sum = 0;
            for(int i=0; i<N; i++ ) {
                if( output[i] != sum ) {
                    fails++;
                }
                sum += input[i];
            }
            std::cerr << "\tex+P\tE="<< fails
                      << "\tS=" << (output[N]==sum?"ok":"ERR" )
                      << "\t"
                      << "\ttime=" << (ms/its) << "ms"
                      << "\tspeedup=" << (ref/(ms/its)) <<"X.\n";
        }


        // exclusive scan with padded and written sum
        if(1) {
            unsigned int* zerocopy;
            hipHostAlloc( &zerocopy, sizeof(unsigned int), hipHostMallocMapped );

            unsigned int* zerocopy_d;
            hipHostGetDevicePointer( &zerocopy_d, zerocopy, 0 );
            CHECK_CUDA;

            *zerocopy = 42;

            float ms;
            int its = 100;
            hipEvent_t start, stop;
            hipEventCreate( &start );
            hipEventCreate( &stop );
            hipMemset( output_d, ~0u, sizeof(unsigned int)*output.size() );
            for(int i=0; i<(its+9)/10; i++) {
                sist::scan::exclusiveScanPadWithSumWriteSum( output_d,
                                                       zerocopy_d,
                                                       scratch_d,
                                                       input_d,
                                                       N );
            }
            hipEventRecord( start, 0 );
            for(int i=0; i<its; i++) {
                sist::scan::exclusiveScanPadWithSumWriteSum( output_d,
                                                       zerocopy_d,
                                                       scratch_d,
                                                       input_d,
                                                       N );
            }
            hipEventRecord( stop );
            hipEventSynchronize( stop );
            hipMemcpy( output.data(), output_d, sizeof(unsigned int)*(output.size()), hipMemcpyDeviceToHost  );
            hipEventElapsedTime( &ms, start, stop );
            CHECK_CUDA;

            int fails = 0;
            unsigned int sum = 0;
            for(int i=0; i<N; i++ ) {
                if( output[i] != sum ) {
                    fails++;
                }
                sum += input[i];
            }
            std::cerr << "\tex+P+S\tE="<< fails
                      << "\tS=" << (output[N]==sum?"ok":"ERR" )
                      << "\tZ=" << ((*zerocopy == sum) ? "ok":"ERR" )
                      << "\ttime=" << (ms/its) << "ms"
                      << "\tspeedup=" << (ref/(ms/its)) <<"X.\n";
            hipHostFree( zerocopy );
        }



    }

    return EXIT_SUCCESS;

}

